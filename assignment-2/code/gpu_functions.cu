#include "hip/hip_runtime.h"
/* 
Code Purpose: 
Author: 
Date: 
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <CUDA_functions.h>

// Prototype Functions 
// __global__ simulation

// Surface Intialization
surface<void, cudaSurfaceType2D> intial_surf;
surface<void, cudaSurfaceType2D> surf;

__host__ void CUDA_errorcheck(){
    hipError_t error = hipGetLastError();
    if (error != hipSuccess){
        printf("CUDA Error: %s \n", hipGetErrorString(error));
        exit(1);
    }
}

// Kernal Function for Radiator Simulation 
__global__ void radiator_sim(float *currentRadiator_gpu, float *passedRadiator_gpu, int n, int m, int iterations)
{
    float values[5]; // Array to hold the values of the neighbors and performing temperature weighting.
    
    // Get the thread ID
    int blockID = blockIdx.x * m; // Block IDs for kernal to process 
    int maxID = blockID + m; // Max ID for kernal to process

    if (maxID > m) // If the max ID is greater than the total number of threads, set it to the total number of threads
    {
        maxID = m;
    }

    // Loop through the iterations 
    for (int i =0; i < maxID; i++){
        int offset = i * m; // Offset for the threads to process
        // Loop through the threads 
        for (int j = blockID; j < maxID; j++)
        {
            j = j + offset; // Add the offset to the thread ID
            // Get the values of the neighbors
            values[0] = currentRadiator_gpu[j - 2]; // Leftmost 
            values[1] = currentRadiator_gpu[j - 1]; // Left
            values[2] = currentRadiator_gpu[j]; // Center
            values[3] = currentRadiator_gpu[j + 1]; // Right
            values[4] = currentRadiator_gpu[j + 2]; // Rightmost

            // Perform the temperature weighting
            currentRadiator_gpu[i * n + j] = (values[0] * 1.65) + (values[1] * 1.35) + (values[2]) + (values[3] * 0.65) + (values[4] * 0.35);
            // printf("row, col: %d, %d \n Rad Value: %f", i, j, currentRadiator_gpu[i * n + j]); // for debugging

            __syncthreads(); // Sync the threads
        }
    }

}


__global__ void printAvg(float *row_temperatures, int m, int iterations) {
	printf("GPU Average Temperatures for timestep %d:", iterations);
	for (int i = 0; i < m; i++) {
		if (i % 9 == 0) printf("\n");
		printf(" %f", row_temperatures[i]);
	}
}

// GPU Reduction Function
__global__ void reduce_matrix_gpu(float *input_vector, float *output_vector, int n, int m)
{
    // Get the thread ID
    int blockID = blockIdx.x * m; // Block IDs for kernal to process 
    int maxID = blockID + m; // Max ID for kernal to process

    if (maxID > m) // If the max ID is greater than the total number of threads, set it to the total number of threads
    {
        maxID = m;
    }

    // Loop through the threads 
    for (int i =0; i < maxID; i++){
        for (int j = threadIdx.x; j < n; j+=blockDim.x)
        {
            output_vector[i] += input_vector[i * n + j]; // Add the values to the output vector

            __syncthreads(); // Sync the threads
        }
    }
}

// Main Function to simulate on the GPU 

// __global__ void reduce_matrix_gpu(float *input_vector, float *output_vector, int n, int m)
// {
//     int j = blockIdx.x * blockDim.x + threadIdx.x;

//     if (j < m) {
//         output_vector[j] = 0.0f;
//         for (int i = 0; i < n; i++) {
//             output_vector[j] += input_vector[i * m + j];
//         }
//     }
//     __syncthreads();
// }


// Main Function to simulate on the GPU 
__host__ extern int radiator_gpu(float *temperatures, float *temperatures_gpu, float **radiator, int n, int m, int iterations, int threads, int timing, int verbose, int M_iterations){

    if (verbose){
        printf("Beginning GPU Simulation");
    }

    // Initialize the variables
    float *row_temperatures_gpu, *currentRadiator_gpu, *passedRadiator_gpu; // *output_vector_gpu, *output_vector_cpu;
    int n_counts = 32; int m_counts = 32; 

    while ((m/threads) > n_counts){
        n_counts = n_counts * 2; // To balance the workload between threads by increasing the number of threads being used until the workload per thread is sufficiently small.
    }

    // Time Keeping Variables
    hipEvent_t start, stop, runtime_start, runtime_stop;
    float time, runtime;
    hipEventCreate(&start); hipEventCreate(&stop); 
    hipEventCreate(&runtime_start); hipEventCreate(&runtime_stop);

    if (timing) {
        hipEventRecord(start, 0); // Start the timer
    }

    hipMalloc((void**)&row_temperatures_gpu, sizeof(float) * m); // Allocate memory for the temperatures
    CUDA_errorcheck(); // Check for errors

    if (verbose){
        printf("Allocating memory for GPU arrays!");
    }

    // Allocate memory for the GPU arrays
    hipMalloc((void**)&currentRadiator_gpu, sizeof(float) * m * n); // Allocate memory for the current radiator
    hipMalloc((void**)&passedRadiator_gpu, sizeof(float) * m * n); // Allocate memory for the passed radiator
    CUDA_errorcheck(); // Check for errors
    hipDeviceSynchronize(); // Sync the threads

    if (timing) {
        hipEventRecord(stop, 0); // Stop the timer
        hipEventSynchronize(stop); // Sync the threads
        hipEventElapsedTime(&time, start, stop); // Get the elapsed time
        printf("Time to allocate memory for GPU arrays: %f ms", time);
    }

    // Copy the data from the CPU to the GPU
    hipEventRecord(start, 0); // Start the timer

    hipMemcpy(currentRadiator_gpu, *radiator, sizeof(float) * m * n, hipMemcpyHostToDevice); // Copy the current radiator to the GPU
    hipMemcpy(passedRadiator_gpu, *radiator, sizeof(float) * m * n, hipMemcpyHostToDevice); // Copy the passed radiator to the GPU
    CUDA_errorcheck(); // Check for errors
    hipDeviceSynchronize(); // Sync the threads

    if (timing) {
        hipEventRecord(stop, 0); // Stop the timer
        hipEventSynchronize(stop); // Sync the threads
        hipEventElapsedTime(&time, start, stop); // Get the elapsed time
        printf("Time to copy data from CPU to GPU: %f ms", time);
    }

    // GPU Block and Grid Configuration
    if (verbose == 1){
        printf("Configuring GPU Blocks and Grids!");
    }
    dim3 dimBlock(threads); // Number of threads per block
    dim3 dimGrid((m/threads) + 1); // Number of blocks per grid

    if (verbose == 1){
        printf("dimBlock: %d, dimGrid: %d", dimBlock.x, dimGrid.x);
    }
    CUDA_errorcheck(); // Check for errors
    hipDeviceSynchronize(); // Sync the threads

    // Run the simulation
    if (verbose == 1){
        printf("Running the simulation!");
    }
    if (timing == 1){
        hipEventRecord(runtime_start, 0); // Start the timer
    }
    // iterate through the number of iterations
    for (int i = 0; i < iterations; i++)
    {
        radiator_sim<<<dimGrid, dimBlock>>>(currentRadiator_gpu, passedRadiator_gpu, n, m, iterations); // Run the simulation
        CUDA_errorcheck(); // Check for errors
        hipDeviceSynchronize(); // Sync the threads

        // Swap the arrays
        float *temp = currentRadiator_gpu;
        currentRadiator_gpu = passedRadiator_gpu;
        passedRadiator_gpu = temp;

        // Printing outputs after a set number of iterations
        if (M_iterations != 0 && i % M_iterations == 0)
        {
            // Average the temperatures
            reduce_matrix_gpu<<<dimGrid, dimBlock>>>(passedRadiator_gpu, row_temperatures_gpu, n, m);
            // print the average temperatures
            printAvg<<<1,1>>>(row_temperatures_gpu, n, iterations);
            hipMemcpy(row_temperatures_gpu, temperatures_gpu, sizeof(float) * m, hipMemcpyDeviceToHost); // Copy the temperatures to the CPU
            CUDA_errorcheck(); // Check for errors
            hipDeviceSynchronize(); // Sync the threads
        }
    }

    reduce_matrix_gpu<<<dimGrid, dimBlock>>>(passedRadiator_gpu, row_temperatures_gpu, n, m);
    CUDA_errorcheck(); // Check for errors

    hipMemcpy(&temperatures[0], temperatures_gpu, sizeof(float) * m, hipMemcpyDeviceToHost); // Copy the temperatures to the CPU
    CUDA_errorcheck(); // Check for errors

    // final and total time keeping 
    if (timing == 1){
        hipEventRecord(runtime_stop, 0); // Stop the timer
        hipEventSynchronize(runtime_stop); // Sync the threads
        hipEventElapsedTime(&runtime, runtime_start, runtime_stop); // Get the elapsed time
        printf("Total time to run the simulation: %f ms", runtime);
    }

    // Free the memory
    hipFree(currentRadiator_gpu); // Free the memory for the current radiator
    hipFree(passedRadiator_gpu); // Free the memory for the passed radiator
    hipFree(temperatures_gpu); // Free the memory for the temperatures
    CUDA_errorcheck(); // Check for errors

    return 0; 
}