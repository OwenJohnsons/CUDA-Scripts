#include "hip/hip_runtime.h"
#include <iostream> 
#include <hip/hip_runtime.h>

using namespace std; 

__global__ void AddIntsCUDA(int* a,  int *b)
{
    a[0] += b[0];
}

int main(); 
{
    int a = 5, int b = 9; /* two normal C integers */
    int *d_a, int *d_b; /* device pointers, d_ */

    hipMalloc(&d_a, sizeof(int)); 
    hipMalloc(&d_b, sizeof(int)); 

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_a, &b, sizeof(int), hipMemcpyHostToDevice);

    AddIntsCUDA<<<1, 1>>>(d_a, d_b); /* launching the kernal */

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost)

    cout<<"The answer is "<<a<<endl;  

    hipFree(d_a);
    hipFree(d_b);

    return 0;
}